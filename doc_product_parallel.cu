
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8
#define THREADS_PER_BLOCK 4
#define BLOCKS (N / THREADS_PER_BLOCK)

__global__ void dot_product(int *a, int *b, int *res)
{
  __shared__ int temp[THREADS_PER_BLOCK];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  temp[threadIdx.x] = a[idx] * b[idx];

  __syncthreads();

  if(0 == threadIdx.x) 
  {
    int sum = 0;
    for(int i = 0; i < THREADS_PER_BLOCK; i++)
      sum += temp[i];
    /* synchronise adding result to sum because
     * *res += sum; can result in a race condition
     */
    atomicAdd(res, sum);
  }
}

void random_ints(int *arr, int n) 
{
  int i;
  for(i = 0; i < n; i++)
    arr[i] = i; /*rand();*/
}

void print_arr(int *arr, int n) 
{
  int i, last;
  for(i = 0, last = n -1; i < last; i++)
    printf("%i,", arr[i]);
  printf("%i\n", arr[last]);
}

int main(void) 
{
  int *a, *b, *res;
  int *dev_a, *dev_b, *dev_res;
  int size = N * sizeof(int);
  
  hipMalloc((void**) &dev_a, size);
  hipMalloc((void**) &dev_b, size);
  hipMalloc((void**) &dev_res, sizeof(int));

  a = (int*) malloc(size);
  b = (int*) malloc(size);
  res = (int*) malloc(sizeof(int));

  random_ints(a, N);
  random_ints(b, N);
  *res = 0;
  
  /* copy dev_a, dev_b to the device */
  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_res, res, sizeof(int), hipMemcpyHostToDevice);

  /* launch device_add kernel with M blocks of N threads. */
  dot_product<<<BLOCKS, THREADS_PER_BLOCK>>>(dev_a, dev_b, dev_res);
  
  /* copy the device result (dev_res) back to res (on host) */
  hipMemcpy(res, dev_res, sizeof(int), hipMemcpyDeviceToHost);

  print_arr(a, N);
  print_arr(b, N);
  printf("result = %i\n", *res);
  
  free(a);
  free(b);
  free(res);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_res);

  return 0;
}


