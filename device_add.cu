
#include <hip/hip_runtime.h>
#include <stdio.h>

/* add a and b on the device.
 * uses pointers because function will run on the device.
 */
__global__ void device_add(int *a, int *b, int *res) 
{
  *res = *a + *b;
}

int main(void)
{
  int a, b, res; /* host copies */
  int *dev_a, *dev_b, *dev_res; /* device copies */
  int size = sizeof(int); /* space needed for an integer */

  /* allocate device copies of a, b, c */
  hipMalloc((void**)&dev_a, size);
  hipMalloc((void**)&dev_b, size);
  hipMalloc((void**)&dev_res, size);

  a = 2;
  b = 7;

  /* copy the inputs to the device */
  hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

  /* launch device_add() on the gpu. */
  device_add<<<1, 1>>>(dev_a, dev_b, dev_res);

  /* copy the device result dev_res, back to the host copy, res. */
  hipMemcpy(&res, dev_res, size, hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_res);

  printf("result: %i\n", res);

  return 0;
}
