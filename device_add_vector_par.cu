
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void device_add(int *a, int *b, int *res) 
{
  res[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 8

void random_ints(int *arr, int n) 
{
  int i;
  for(i = 0; i < n; i++)
    arr[i] = rand();
}

void print_arr(int *arr, int n) 
{
  int i, last;
  for(i = 0, last = n -1; i < last; i++)
    printf("%i,", arr[i]);
  printf("%i\n", arr[last]);
}

int main(void) 
{
  int *a, *b, *res;
  int *dev_a, *dev_b, *dev_res;
  int size = N * sizeof(int);
  
  hipMalloc((void**) &dev_a, size);
  hipMalloc((void**) &dev_b, size);
  hipMalloc((void**) &dev_res, size);

  a = (int*) malloc(size);
  b = (int*) malloc(size);
  res = (int*) malloc(size);
  
  random_ints(a, N);
  random_ints(b, N);
  
  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  device_add<<<N, 1>>>(dev_a, dev_b, dev_res);
  
  hipMemcpy(res, dev_res, size, hipMemcpyDeviceToHost);

  print_arr(res, N);

  free(a);
  free(b);
  free(res);
  
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_res);

  return 0;
}


